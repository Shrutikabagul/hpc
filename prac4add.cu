// cuda addition

#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

// --------------------------------------
// CUDA Kernel for Vector Addition
__global__ void vectorAdd(int* A, int* B, int* C, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n)
        C[i] = A[i] + B[i];
}

// --------------------------------------
int main() {
    int n;
    cout << "Enter size of vectors (e.g. 1024): ";
    cin >> n;

    // Host memory allocation for vectors
    int* h_A = new int[n];
    int* h_B = new int[n];
    int* h_C = new int[n];

    // Initialize vectors
    for (int i = 0; i < n; i++) {
        h_A[i] = i;
        h_B[i] = 2 * i;
    }

    // Device memory allocation
    int *d_A, *d_B, *d_C;
    hipMalloc(&d_A, n * sizeof(int));
    hipMalloc(&d_B, n * sizeof(int));
    hipMalloc(&d_C, n * sizeof(int));

    // Copy inputs to device
    hipMemcpy(d_A, h_A, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, n * sizeof(int), hipMemcpyHostToDevice);

    // Launch vector addition kernel
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    vectorAdd<<<gridSize, blockSize>>>(d_A, d_B, d_C, n);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(h_C, d_C, n * sizeof(int), hipMemcpyDeviceToHost);

    cout << "\nVector Addition Result (first 10 values): ";
    for (int i = 0; i < min(10, n); i++) {
        cout << h_C[i] << " ";
    }
    cout << endl;

    // Free memory
    delete[] h_A; delete[] h_B; delete[] h_C;
    hipFree(d_A); hipFree(d_B); hipFree(d_C);

    return 0;
}
